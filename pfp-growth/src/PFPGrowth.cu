//
// Created by rafael on 01/09/18.
//
#include "cudaHeaders.h"
#include "Kernel.h"
#include "PFPArray.h"
#include "PFPGrowth.cu.h"
#include "PFPArray.h"
#include "PFPArray.h"
#include "../include/PFPArray.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

PFPGrowth::PFPGrowth(ArrayMap *arrayMap, Elo *eloMap, size_t arrayMapSize, size_t eloPosMapSize) {
    ArrayMap *device_ArrayMap;
    Elo *device_EloMap;
    EloVector *device_pointer_elo_kx, *host_pointer_elo_kx, *data_host_pointer_elo_kx;

    data_host_pointer_elo_kx = (EloVector *)malloc(sizeof(EloVector)*eloPosMapSize);
    for (int j = 0; j < eloPosMapSize; ++j) {
        data_host_pointer_elo_kx[j].eloArray=(Elo *)malloc(sizeof(Elo)*eloPosMapSize);
    }
    data_host_pointer_elo_kx->eloArray=eloMap;
    data_host_pointer_elo_kx->size=eloPosMapSize;

    host_pointer_elo_kx = (EloVector*)malloc(eloPosMapSize * sizeof(EloVector));
    memcpy(host_pointer_elo_kx, data_host_pointer_elo_kx, eloPosMapSize * sizeof(EloVector));

    for (int i=0; i<eloPosMapSize; i++){
        hipMalloc(&(host_pointer_elo_kx[i].eloArray), eloPosMapSize*sizeof(Elo));
        hipMemcpy(host_pointer_elo_kx[i].eloArray, data_host_pointer_elo_kx[i].eloArray, eloPosMapSize*sizeof(Elo), hipMemcpyHostToDevice);
    }

    hipMalloc((void **)&device_pointer_elo_kx, sizeof(EloVector)*eloPosMapSize);
    hipMemcpy(device_pointer_elo_kx,host_pointer_elo_kx,sizeof(EloVector)*eloPosMapSize,hipMemcpyHostToDevice);

    gpuErrchk(hipMalloc((void **) &device_ArrayMap, sizeof(ArrayMap) * arrayMapSize));
    gpuErrchk(hipMemcpy(device_ArrayMap, arrayMap, sizeof(ArrayMap) * arrayMapSize, hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc((void **) &device_EloMap, sizeof(Elo) * eloPosMapSize));
    gpuErrchk(hipMemcpy(device_EloMap, eloMap, sizeof(Elo) * eloPosMapSize, hipMemcpyHostToDevice));

    pfp_growth << < 1,eloPosMapSize >>>
            (device_pointer_elo_kx,0,
                    device_ArrayMap,
                    arrayMapSize);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());


//    for (int l = 0; l < eloPosMapSize ; ++l) {
//        hipMemcpy(host_pointer_elo_kx[l],
//                   device_pointer_elo_kx[l],
//                   sizeof(Elo)*eloPosMapSize,
//                   hipMemcpyDeviceToHost);
//
//    }

    hipFree(device_EloMap);
    hipFree(device_ArrayMap);


}
