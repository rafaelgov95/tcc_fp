//
// Created by rafael on 01/09/18.
//
#include "cudaHeaders.h"
#include "Kernel.h"
#include "PFPArray.h"
#include "PFPGrowth.cu.h"
#include "PFPArray.h"
#include "PFPArray.h"
#include "../include/PFPArray.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
    {
        if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
PFPGrowth::PFPGrowth(gpuArrayMap *arrayMap, gpuEloMap *eloMap,size_t arrayMapSize,size_t eloPosMapSize) {
    gpuArrayMap *device_ArrayMap;
    gpuEloMap *device_EloMap;
    EloGrid **device_elo_grid;


    gpuErrchk(hipMalloc((void **) &device_ArrayMap, sizeof(gpuArrayMap)*arrayMapSize));
    gpuErrchk(hipMemcpy(device_ArrayMap, arrayMap, sizeof(gpuArrayMap)*arrayMapSize, hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc((void **) &device_EloMap, sizeof(gpuEloMap)*eloPosMapSize));
    gpuErrchk(hipMemcpy(device_EloMap, eloMap, sizeof(gpuArrayMap)*eloPosMapSize, hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc((void **) &device_elo_grid, sizeof(EloGrid)*eloPosMapSize));

    run<<<1,eloPosMapSize>>>(device_elo_grid,device_ArrayMap,device_EloMap,arrayMapSize,eloPosMapSize);

    gpuErrchk( hipPeekAtLastError());
    gpuErrchk( hipDeviceSynchronize() );
    hipFree(device_EloMap);
    hipFree(device_ArrayMap);
    hipFree(device_elo_grid);
}
