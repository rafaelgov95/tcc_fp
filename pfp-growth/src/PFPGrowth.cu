//
// Created by rafael on 01/09/18.
//
#include "cudaHeaders.h"
#include "Kernel.h"
#include "PFPArray.h"
#include "PFPGrowth.cu.h"
#include "PFPArray.h"
#include "PFPArray.h"
#include "../include/PFPArray.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

PFPGrowth::PFPGrowth(ArrayMap *arrayMap, Elo *eloMap, size_t arrayMapSize, size_t eloPosMapSize) {
    ArrayMap *device_ArrayMap;
    EloVector *device_pointer_elo_vector, *host_elos_vector_and_memory_pointer_elos, *data_host_elos_vector;
    Elo *host_elos[eloPosMapSize];
    int *deviceEloVectorSize;
    int hostEloVectorSize=1;


    data_host_elos_vector = (EloVector *)malloc(sizeof(EloVector)*eloPosMapSize);
    for (int j = 0; j < eloPosMapSize; ++j) {
        data_host_elos_vector[j].eloArray=(Elo *)malloc(sizeof(Elo)*eloPosMapSize);
    }
    data_host_elos_vector[0].eloArray=eloMap;
    data_host_elos_vector[0].size=eloPosMapSize;

    host_elos_vector_and_memory_pointer_elos = (EloVector*)malloc(eloPosMapSize * sizeof(EloVector));
    memcpy(host_elos_vector_and_memory_pointer_elos, data_host_elos_vector, eloPosMapSize * sizeof(EloVector));

    for (int i=0; i<eloPosMapSize; i++){
        hipMalloc(&(host_elos_vector_and_memory_pointer_elos[i].eloArray), eloPosMapSize*2*sizeof(Elo));
        hipMemcpy(host_elos_vector_and_memory_pointer_elos[i].eloArray, data_host_elos_vector[i].eloArray, eloPosMapSize*sizeof(Elo), hipMemcpyHostToDevice);
    }
    hipMalloc((void **)&device_pointer_elo_vector, sizeof(EloVector)*eloPosMapSize);
    hipMemcpy(device_pointer_elo_vector,host_elos_vector_and_memory_pointer_elos,sizeof(EloVector)*eloPosMapSize,hipMemcpyHostToDevice);


    gpuErrchk(hipMalloc((void **) &device_ArrayMap, sizeof(ArrayMap) * arrayMapSize));

    gpuErrchk(hipMalloc((void **) &deviceEloVectorSize, sizeof(int)));

    gpuErrchk(hipMemcpy(device_ArrayMap, arrayMap, sizeof(ArrayMap) * arrayMapSize, hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(deviceEloVectorSize,&hostEloVectorSize, sizeof(int), hipMemcpyHostToDevice));

    pfp_growth << < 1,eloPosMapSize,50*sizeof(Elo)>>>
                  (device_pointer_elo_vector,
                    deviceEloVectorSize,
                    device_ArrayMap,
                    arrayMapSize);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    for(int i =0;i<eloPosMapSize;++i) {
        host_elos[i] = (Elo *) malloc(eloPosMapSize*2* sizeof(Elo)); //Tamanho ficou pequeno para o final
    }

    gpuErrchk(hipMemcpy(host_elos_vector_and_memory_pointer_elos,device_pointer_elo_vector,sizeof(EloVector)*eloPosMapSize,hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(&hostEloVectorSize,deviceEloVectorSize,sizeof(int),hipMemcpyDeviceToHost));


    for(int i =0;i<eloPosMapSize;++i){
        gpuErrchk(hipMemcpy(host_elos[i],host_elos_vector_and_memory_pointer_elos[i].eloArray,sizeof(Elo)*eloPosMapSize*2,hipMemcpyDeviceToHost)); //Tamanho ficou pequeno para o final

    }

    printf("Total de Gerações de Frequência %d\n",hostEloVectorSize+1);
    for (int k = 0; k <=hostEloVectorSize; ++k) {
        for (int j = 0; j <host_elos_vector_and_memory_pointer_elos[k].size; ++j) {
            printf("%s;%d;%d \n",host_elos[k][j].ItemId,host_elos[k][j].indexArrayMap,host_elos[k][j].suporte);
        }
    }


    hipFree(device_ArrayMap);
    hipFree(deviceEloVectorSize);
    hipFree(device_pointer_elo_vector);
    hipFree(host_elos_vector_and_memory_pointer_elos->eloArray);
}
