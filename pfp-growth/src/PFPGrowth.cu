//
// Created by rafael on 01/09/18.
//
#include "cudaHeaders.h"
#include "Kernel.h"
#include "PFPArray.h"
#include "PFPGrowth.cu.h"
#include "PFPArray.h"
#include "PFPArray.h"
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
PFPGrowth::PFPGrowth(gpuArrayMap *arrayMap, gpuEloMap *eloMap) {
    gpuArrayMap *device_ArrayMap;
    gpuErrchk(hipMalloc((void **) &device_ArrayMap, sizeof(arrayMap)));
    gpuErrchk(hipMemcpy(device_ArrayMap, arrayMap, sizeof(arrayMap), hipMemcpyHostToDevice));
    AlgoritmoI<<<1,12>>>(device_ArrayMap,eloMap);
    gpuErrchk( hipPeekAtLastError());
    gpuErrchk( hipDeviceSynchronize() );
}
