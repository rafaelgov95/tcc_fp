//
// Created by rafael on 01/09/18.
//
#include "cudaHeaders.h"
#include "Kernel.h"
#include "PFPArray.h"
#include "PFPGrowth.cu.h"
#include "PFPArray.h"
#include "PFPArray.h"
#include "../include/PFPArray.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

PFPGrowth::PFPGrowth(ArrayMap *arrayMap, Elo *eloMap, size_t arrayMapSize, size_t eloPosMapSize) {
    ArrayMap *device_ArrayMap;
    Elo *device_EloMap;
    Elo *device_pointer_elo_kx[eloPosMapSize], *host_pointer_elo_kx[eloPosMapSize];
    int *device_int_array;

    hipMalloc((void**) &device_int_array, sizeof(int)*eloPosMapSize);
    hipMemset (device_int_array , 0 , eloPosMapSize * sizeof ( int ));
    for (int i = 0; i < eloPosMapSize ; ++i) {
         host_pointer_elo_kx[i]=(Elo*)malloc(sizeof(Elo) * eloPosMapSize);
   }
   for (int i = 0; i < eloPosMapSize ; ++i) {
        hipMalloc((void **) &device_pointer_elo_kx[i], sizeof(Elo) * eloPosMapSize);
    }

    for (int j = 0; j < eloPosMapSize; ++j) {
        hipMemcpy(device_pointer_elo_kx[j],host_pointer_elo_kx[j],sizeof(Elo) *eloPosMapSize,hipMemcpyHostToDevice);
    }

    gpuErrchk(hipMalloc((void **) &device_ArrayMap, sizeof(ArrayMap) * arrayMapSize));
    gpuErrchk(hipMemcpy(device_ArrayMap, arrayMap, sizeof(ArrayMap) * arrayMapSize, hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc((void **) &device_EloMap, sizeof(Elo) * eloPosMapSize));
    gpuErrchk(hipMemcpy(device_EloMap, eloMap, sizeof(Elo) * eloPosMapSize, hipMemcpyHostToDevice));

    run << < 1,eloPosMapSize >>>
            (device_pointer_elo_kx,
                    device_int_array,
                    device_ArrayMap,
                    device_EloMap,
                    arrayMapSize,
                    eloPosMapSize);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());


//    for (int l = 0; l < eloPosMapSize ; ++l) {
//        hipMemcpy(host_pointer_elo_kx[l],
//                   device_pointer_elo_kx[l],
//                   sizeof(Elo)*eloPosMapSize,
//                   hipMemcpyDeviceToHost);
//
//    }

    hipFree(device_EloMap);
    hipFree(device_ArrayMap);


}
