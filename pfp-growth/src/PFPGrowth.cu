//
// Created by rafael on 01/09/18.
//
#include "cudaHeaders.h"
#include "Kernel.h"
#include "PFPArray.h"
#include "PFPGrowth.cu.h"
#include "PFPArray.h"
#include "PFPArray.h"
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
    {
        if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
PFPGrowth::PFPGrowth(gpuArrayMap *arrayMap, gpuEloMap *eloMap,size_t arrayMapSize,size_t eloPosMapSize) {
    gpuArrayMap *device_ArrayMap;
    gpuEloMap *device_EloMap;
    gpuEloMap **device_elo_frequencias;

    gpuErrchk(hipMalloc((void **) &device_ArrayMap, sizeof(gpuArrayMap)*arrayMapSize));
    gpuErrchk(hipMemcpy(device_ArrayMap, arrayMap, sizeof(gpuArrayMap)*arrayMapSize, hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc((void **) &device_EloMap, sizeof(gpuEloMap)*eloPosMapSize));
    gpuErrchk(hipMemcpy(device_EloMap, eloMap, sizeof(gpuArrayMap)*eloPosMapSize, hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc((void **) &device_elo_frequencias, sizeof(gpuEloMap)*eloPosMapSize*eloPosMapSize));


    AlgoritmoI<<<1,arrayMapSize-1>>>(device_elo_frequencias,device_ArrayMap,device_EloMap,arrayMapSize,0);
    gpuErrchk( hipPeekAtLastError());
    gpuErrchk( hipDeviceSynchronize() );
    hipFree(device_EloMap);
    hipFree(device_ArrayMap);
}
