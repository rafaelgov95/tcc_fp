//
// Created by rafael on 01/09/18.
//
#include "cudaHeaders.h"
#include "Kernel.h"
#include "PFPArray.h"
#include "PFPGrowth.cu.h"
#include "PFPArray.h"
#include "../include/PFPArray.h"
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
PFPGrowth::PFPGrowth(std::vector <PFPArrayMap> arrayMap) {

    gpuArrayMap *host_ArrayMap = (gpuArrayMap *) malloc(sizeof(gpuArrayMap) * arrayMap.size());
    for (auto it = arrayMap.begin(); it != arrayMap.end(); ++it) {
        int index = std::distance(arrayMap.begin(), it);
        host_ArrayMap[index].suporte = (*it).suporte;
        host_ArrayMap[index].indexP = (*it).indexP;
        std::strcpy(host_ArrayMap[index].ItemId, (*it).ItemId->item.c_str());
//        std::cout<<a[index].ItemId<<std::endl;
    }

    int size = sizeof(gpuArrayMap) * arrayMap.size();
    gpuArrayMap *device_ArrayMap;
    gpuErrchk(hipMalloc((void **) &device_ArrayMap, size));
    gpuErrchk(hipMemcpy(device_ArrayMap, host_ArrayMap, size, hipMemcpyHostToDevice));
    std::cout<<host_ArrayMap[0].indexP<<std::endl;

    AlgoritmoI<<<1,12>>>(device_ArrayMap);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}
