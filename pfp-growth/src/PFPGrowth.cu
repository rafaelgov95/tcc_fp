//
// Created by rafael on 01/09/18.
//
#include "cudaHeaders.h"
#include "Kernel.h"
#include "PFPArray.h"
#include "PFPGrowth.cu.h"
#include "PFPArray.h"
#include "PFPArray.h"
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
PFPGrowth::PFPGrowth(gpuArrayMap *arrayMap, gpuEloMap *eloMap,size_t arrayMapSize,size_t eloPosMapSize) {
    gpuArrayMap *device_ArrayMap;
    gpuEloMap *device_EloMap;
    gpuErrchk(hipMalloc((void **) &device_ArrayMap, sizeof(gpuArrayMap)*arrayMapSize));
    gpuErrchk(hipMemcpy(device_ArrayMap, arrayMap, sizeof(gpuArrayMap)*arrayMapSize, hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc((void **) &device_EloMap, sizeof(gpuEloMap)*eloPosMapSize));
    gpuErrchk(hipMemcpy(device_EloMap, eloMap, sizeof(gpuArrayMap)*eloPosMapSize, hipMemcpyHostToDevice));

    AlgoritmoI<<<1,12>>>(device_ArrayMap,device_EloMap);
    gpuErrchk( hipPeekAtLastError());
    gpuErrchk( hipDeviceSynchronize() );
}
