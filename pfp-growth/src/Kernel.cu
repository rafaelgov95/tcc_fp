#include "hip/hip_runtime.h"
//
// Created by rafael on 20/08/18.
//

#include <cudaHeaders.h>
#include "Kernel.h"
#include "PFPTree.h"
#include "PFPArray.h"
#include <hip/hip_runtime_api.h>
#include <cstdio>
#include "hip/hip_runtime.h"
#include "../include/PFPArray.h"
#include "../../../../../../usr/include/form.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

typedef struct {
    Elo elo;
    int size;
}SetMap;

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
__device__ char *counter1;
__device__ int counter2;
__device__ int index_elo_setmap;
__shared__ Elo elo[256];
__device__ int index_elo_put;


__device__ int compare(char* String_1, char* String_2)
{
    char TempChar_1,
            TempChar_2;

    do
    {
        TempChar_1 = *String_1++;
        TempChar_2 = *String_2++;
    } while(TempChar_1 && TempChar_1 == TempChar_2);

    return TempChar_1 - TempChar_2;
}

__device__ bool my_strcmp( char *array1, char *array2) {
    int i = 0;
    while (array1[i] != '\0') {
        if (array1[i] != array2[i]) {
            return false;
        }
        i++;
    }
    return true;
}
__device__ char *my_strcpy(char *dest, const char *src) {
    int i = 0;
    do {
        dest[i] = src[i];
    } while (src[i++] != 0);
    return dest;
}

__device__ char *my_strcat(char *dest, const char *src) {
    int i = 0;
    while (dest[i] != 0) i++;
    my_strcpy(dest + i, src);
    return dest;
}

__device__ char *my_cpcat(const char *array1, const char *array2, char *src) {
    my_strcat(src, array1);
    my_strcat(src, array2);
    return src;
}

__global__ void frequencia_x( Elo *elo_x,int eloMapSize){
extern __shared__ SetMap setMap[];
int eloSize=0;

memset(setMap,0,sizeof(SetMap)*eloMapSize);
char  valr[23]="";
if(threadIdx.x==0){
   for(int k=0;k<eloMapSize;++k){
          my_strcpy(setMap[k].elo.ItemId," ");
   }

}

//__syncthreads();
if(threadIdx.x==0){

for(int k=0;k<eloMapSize;++k){
        int i=0;
        bool flag= true;
            while(i<eloMapSize && flag){
            if(0==compare(setMap[i].elo.ItemId," ")){
                setMap[i].elo=elo_x[k];
                eloSize++;
                flag =false;
            }else{
                if(0==compare(elo_x[k].ItemId,setMap[i].elo.ItemId)){
                    flag =false;
                    setMap[i].elo.suporte+=elo_x[k].suporte;
                }
            }
            i++;
        }
    }
        atomicAdd(&index_elo_setmap,eloSize);


    }
    if(threadIdx.x==24){
    for(int i =0;i<16;++i ){
        printf("SeMap Thread %d valor MAP %s Suporte %d \n",threadIdx.x,setMap[i].elo.ItemId,setMap[i].elo.suporte);
        }
    }
}


__device__ void pfp_growth(Elo **elo_k1,int *nn,ArrayMap *arrayMap, Elo *eloMap, size_t arrayMapSize, size_t eloMapSize) {

// Algoritmo 1 Begin;

auto indexAtual = threadIdx.x;
    int xxx = 0;
    bool flag = true;
    Elo *Elo_k1 = (Elo *) malloc(sizeof(Elo) * eloMapSize);
    while (flag && (indexAtual + xxx) < eloMapSize) {
        char a[32] = "";
            auto indexParentArrayMap = arrayMap[eloMap[indexAtual + xxx].indexArrayMap].indexP;
            auto indexThreadArrayMap = eloMap[indexAtual].indexArrayMap;
            if (arrayMap[indexThreadArrayMap].indexP != -1 &&
                arrayMap[indexParentArrayMap].indexP != -1) {
                my_cpcat(arrayMap[indexThreadArrayMap].ItemId,
                         arrayMap[indexParentArrayMap].ItemId, Elo_k1[xxx].ItemId);
                Elo_k1[xxx].indexArrayMap = arrayMap[indexParentArrayMap].indexP;
                Elo_k1[xxx].suporte = arrayMap[indexThreadArrayMap].suporte;
            } else {
                my_cpcat(arrayMap[eloMap[indexAtual].indexArrayMap].ItemId,
                         arrayMap[indexParentArrayMap].ItemId, Elo_k1[xxx].ItemId);
                Elo_k1[xxx].indexArrayMap = arrayMap[indexParentArrayMap].indexP;
                Elo_k1[xxx].suporte = arrayMap[indexAtual].suporte;
               flag = false;
            }
            xxx++;

    }
// Algoritmo 1 End;

// Algoritmo 2 Begin;
    for (int i = 0; i < (xxx-1); ++i)
    elo[atomicAdd(&index_elo_put,1)]=Elo_k1[i];

    if (threadIdx.x == eloMapSize-1 ) {
        Elo *elo_x= (Elo *)malloc(sizeof(Elo)*index_elo_put);
        for (int i = 0; i < index_elo_put; ++i){
                    elo_x[i]= elo[i];
        }
        frequencia_x<<<1,index_elo_put,sizeof(SetMap)*index_elo_put*index_elo_put >>>(elo_x,index_elo_put);
        hipDeviceSynchronize();
        index_elo_put=0;
       }
}

__global__ void run(Elo **Elo_k1,int *nn, ArrayMap *arrayMap, Elo *eloMap, size_t ArrayMapSize, size_t eloMapSize) {

    if (threadIdx.x < eloMapSize) {
        pfp_growth(Elo_k1,nn,arrayMap, eloMap, ArrayMapSize, eloMapSize);
    }

}

