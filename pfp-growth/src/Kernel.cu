#include "hip/hip_runtime.h"
//
// Created by rafael on 20/08/18.
//

#include <cudaHeaders.h>
#include "Kernel.h"
#include "PFPTree.h"
#include "PFPArray.h"
#include <hip/hip_runtime_api.h>
#include <cstdio>
#include "hip/hip_runtime.h"
#include "../include/PFPArray.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }


inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__device__ char *my_strcpy(char *dest, const char *src) {
    int i = 0;
    do {
        dest[i] = src[i];
    } while (src[i++] != 0);
    return dest;
}

__device__ char *my_strcat(char *dest, const char *src) {
    int i = 0;
    while (dest[i] != 0) i++;
    my_strcpy(dest + i, src);
    return dest;
}

__device__ char *my_cpcat(const char *array1, const char *array2, char *src) {
    my_strcat(src, array1);
    my_strcat(src, array2);
    return src;
}

__device__ void
reducex_suporte(EloGrid *eloGrid, gpuArrayMap *arrayMap, gpuEloMap *eloMap, size_t arrayMapSize, size_t eloMapSize,
                int elo_k1_map_size, gpuEloMap *Elo_k1) {


    int indexEloMap = eloGrid->size;
    eloGrid->eloMap[indexEloMap].elo = Elo_k1;
    eloGrid->eloMap[indexEloMap].size=elo_k1_map_size;

    for (int i = 0; i < eloGrid->eloMap[indexEloMap].size; i++)
        printf("THREAD %d ITEMID %s | IndexArray %d| Suporte %d |\n", threadIdx.x,  eloGrid->eloMap[indexEloMap].elo[i].ItemId,eloGrid->eloMap[indexEloMap].elo[i].indexArrayMap,eloGrid->eloMap[indexEloMap].elo[i].suporte);
    eloGrid->size=+eloGrid->size;
}


__device__ void
geracao_candidato(EloGrid **Elo_Grid, gpuArrayMap *arrayMap, gpuEloMap *eloMap, size_t arrayMapSize, size_t eloMapSize) {
    auto indexAtual = threadIdx.x;
    int xxx = 0;
    bool flag = true;
    gpuEloMap *Elo_k1 = (gpuEloMap *) malloc(sizeof(gpuEloMap) * eloMapSize);
    while (flag && (indexAtual + xxx) < eloMapSize) {
        char a[32] = "";
            auto indexParentArrayMap = arrayMap[eloMap[indexAtual + xxx].indexArrayMap].indexP;
            if (arrayMap[indexParentArrayMap].indexP != -1 &&
                arrayMap[eloMap[indexAtual].indexArrayMap].indexP != -1) {
                my_cpcat(arrayMap[eloMap[indexAtual].indexArrayMap].ItemId,
                         arrayMap[indexParentArrayMap].ItemId, a);
                my_strcpy(Elo_k1[xxx].ItemId, a);
                Elo_k1[xxx].indexArrayMap = arrayMap[eloMap[indexAtual+xxx].indexArrayMap].indexP;
                Elo_k1[xxx].suporte = arrayMap[eloMap[indexAtual+xxx].indexArrayMap].suporte;
            } else {
                my_cpcat(arrayMap[eloMap[indexAtual].indexArrayMap].ItemId,
                         arrayMap[indexParentArrayMap].ItemId, a);
                my_strcpy(Elo_k1[xxx].ItemId, a);
                Elo_k1[xxx].indexArrayMap = arrayMap[indexParentArrayMap].indexP;
                Elo_k1[xxx].suporte = arrayMap[indexAtual].suporte;
               flag = false;
            }
            xxx++;
    }
    reducex_suporte(Elo_Grid[indexAtual], arrayMap, eloMap, arrayMapSize, eloMapSize, xxx - 1, Elo_k1);

}

__global__ void run(EloGrid **Elo_k1, gpuArrayMap *arrayMap, gpuEloMap *eloMap, size_t sizeArrayMap, size_t eloMapSize) {

    if (threadIdx.x < eloMapSize) {
        Elo_k1[threadIdx.x] = (EloGrid *) malloc(sizeof(EloGrid));
        Elo_k1[threadIdx.x]->size = 0;
        Elo_k1[threadIdx.x]->eloMap = (EloMap *) malloc(sizeof(EloMap *)* eloMapSize);
        geracao_candidato(Elo_k1, arrayMap, eloMap, sizeArrayMap, eloMapSize);
    }
//    hipFree(Elo_k1);

}

