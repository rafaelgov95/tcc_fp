#include "hip/hip_runtime.h"
//
// Created by rafael on 20/08/18.
//

#include <cudaHeaders.h>
#include "Kernel.h"
#include "PFPTree.h"
#include "PFPArray.h"
#include <hip/hip_runtime_api.h>
#include <cstdio>
#include "hip/hip_runtime.h"
#include "../include/PFPArray.h"

__global__ void AlgoritmoI(gpuArrayMap *arrayMap, gpuEloMap *eloMap) {
    if (threadIdx.x < 12) {
        printf("ARRAY ITEM: %s | PARENT INDEX %d | SUPORTE %d\n", arrayMap[threadIdx.x].ItemId,
               arrayMap[threadIdx.x].indexP, arrayMap[threadIdx.x].suporte);

    }
    if (threadIdx.x < 11) {
        printf("ELO ITEM: %s |  INDEX ARRAY %d | SUPORTE %d\n", eloMap[threadIdx.x].ItemId,
               eloMap[threadIdx.x].indexArrayMap, eloMap[threadIdx.x].suporte);
    }

}
__global__ void AlgoritmoI(){
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}
