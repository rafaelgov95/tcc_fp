#include "hip/hip_runtime.h"
//
// Created by rafael on 20/08/18.
//

#include <cudaHeaders.h>
#include "Kernel.h"
#include "PFPTree.h"
#include "PFPArray.h"
#include <hip/hip_runtime_api.h>
#include <cstdio>
#include "hip/hip_runtime.h"
#include "../include/PFPArray.h"
#include "../../../../../../usr/include/form.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

typedef struct {
    Elo elo;
    int size;
}SetMap;

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
__device__ char *counter1;
__device__ int inde_new_elo;
__device__ int index_elo_setmap;
__shared__ Elo elo[256];
__device__ int index_elo_put;


__device__ int compare(char* String_1, char* String_2)
{
    char TempChar_1,
            TempChar_2;

    do
    {
        TempChar_1 = *String_1++;
        TempChar_2 = *String_2++;
    } while(TempChar_1 && TempChar_1 == TempChar_2);

    return TempChar_1 - TempChar_2;
}

__device__ bool my_strcmp( char *array1, char *array2) {
    int i = 0;
    while (array1[i] != '\0') {
        if (array1[i] != array2[i]) {
            return false;
        }
        i++;
    }
    return true;
}
__device__ char *my_strcpy(char *dest, const char *src) {
    int i = 0;
    do {
        dest[i] = src[i];
    } while (src[i++] != 0);
    return dest;
}

__device__ char *my_strcat(char *dest, const char *src) {
    int i = 0;
    while (dest[i] != 0) i++;
    my_strcpy(dest + i, src);
    return dest;
}

__device__ char *my_cpcat(const char *array1, const char *array2, char *src) {
    my_strcat(src, array1);
    my_strcat(src, array2);
    return src;
}

__global__ void frequencia_x(Elo **     elo_k1,int rounding,Elo *elo_x,int eloMapSize, int minimo){
extern __shared__ SetMap setMap[];
 __shared__ Elo elo_new_put[70];
int eloSize=0;
//Elo *elo_new_put
//elo_new_put = (Elo *)malloc(sizeof(Elo )* eloMapSize );

memset(elo_new_put,0,sizeof(Elo)*eloMapSize);

memset(setMap,0,sizeof(SetMap)*eloMapSize);
if(threadIdx.x==0){
   for(int k=0;k<eloMapSize;++k){
          my_strcpy(setMap[k].elo.ItemId," ");
   }

}

__syncthreads();
if(threadIdx.x==0){

for(int k=0;k<eloMapSize;++k){
        int i=0;
        bool flag= true;
            while(i<eloMapSize && flag){
            if(0==compare(setMap[i].elo.ItemId," ")){
                setMap[i].elo=elo_x[k];
                eloSize++;
                flag =false;
            }else{
                if(0==compare(elo_x[k].ItemId,setMap[i].elo.ItemId)){
                    flag =false;
                    setMap[i].elo.suporte+=elo_x[k].suporte;
                    }
            }
            i++;
        }
    }
        atomicAdd(&index_elo_setmap,eloSize);
}
__syncthreads();

bool newFlag=true;
int indexSetMap=0;
while(newFlag && indexSetMap < index_elo_setmap){
    if((0==compare(elo_x[threadIdx.x].ItemId,setMap[indexSetMap].elo.ItemId )) && (setMap[indexSetMap].elo.suporte >= minimo)){
        elo_new_put[atomicAdd(&inde_new_elo,1)]=elo_x[threadIdx.x];
        newFlag=false;
    }
    indexSetMap++;
}


    if(threadIdx.x==eloMapSize-1){
        for(int i =0;i<index_elo_setmap;++i ){
            printf("SetMAP Thread %d valor MAP %s Suporte %d \n",threadIdx.x,setMap[i].elo.ItemId,setMap[i].elo.suporte);
        }
        for(int i =0;i<inde_new_elo;++i ){
        printf("Elo_new_PUT Thread %d valor MAP %s Suporte %d \n",threadIdx.x,elo_new_put[i].ItemId,elo_new_put[i].suporte);
        }
//            elo_k1=(Elo **)malloc(sizeof(Elo)*index_elo_put);
            elo_k1[0]=elo_new_put;
//            for (int i = 0; i < index_elo_put; ++i){
//                 elo_x[i]= elo_new_put[i];
//            }
//printf("Elo_new_PUT Thread %d valor MAP %s Suporte %d \n",threadIdx.x,elo_k1.ItemId,elo_k1.suporte;
//        elo_x=elo_new_put;
//        eloMapSize=inde_new_elo;
//        inde_new_elo=0;
    }


//__global__ void runInterno(Elo **Elo_k1,int *nn, ArrayMap *arrayMap, Elo *eloMap, size_t ArrayMapSize, size_t eloMapSize) {


}
__device__ void pfp_growth(Elo **elo_k1,int *nn,ArrayMap *arrayMap, Elo *eloMap, size_t arrayMapSize, size_t eloMapSize) {

// Algoritmo 1 Begin;

auto indexAtual = threadIdx.x;
    int xxx = 0;
    bool flag = true;
    Elo *Elo_k1 = (Elo *) malloc(sizeof(Elo) * eloMapSize);
    while (flag && (indexAtual + xxx) < eloMapSize) {
            auto indexParentArrayMap = arrayMap[eloMap[indexAtual + xxx].indexArrayMap].indexP;
            auto indexThreadArrayMap = eloMap[indexAtual].indexArrayMap;
            if (arrayMap[indexThreadArrayMap].indexP != -1 &&
                arrayMap[indexParentArrayMap].indexP != -1) {
                my_cpcat(arrayMap[indexThreadArrayMap].ItemId,
                         arrayMap[indexParentArrayMap].ItemId, Elo_k1[xxx].ItemId);
                Elo_k1[xxx].indexArrayMap = arrayMap[indexParentArrayMap].indexP;
                Elo_k1[xxx].suporte = arrayMap[indexThreadArrayMap].suporte;
            } else {
                my_cpcat(arrayMap[eloMap[indexAtual].indexArrayMap].ItemId,
                         arrayMap[indexParentArrayMap].ItemId, Elo_k1[xxx].ItemId);
                Elo_k1[xxx].indexArrayMap = arrayMap[indexParentArrayMap].indexP;
                Elo_k1[xxx].suporte = arrayMap[indexAtual].suporte;
               flag = false;
            }
            xxx++;

    }
// Algoritmo 1 End;

// Algoritmo 2 Begin;
    for (int i = 0; i < (xxx-1); ++i)
    elo[atomicAdd(&index_elo_put,1)]=Elo_k1[i];

    if (threadIdx.x == eloMapSize-1 ) {
        Elo *elo_x= (Elo *)malloc(sizeof(Elo)*index_elo_put);
        for (int i = 0; i < index_elo_put; ++i){
                    elo_x[i]= elo[i];
        }
        printf("SUPER IMPORTANTE ANTES %d\n",index_elo_put);
        frequencia_x<<<1,index_elo_put,sizeof(SetMap)*index_elo_put>>>(elo_k1,1,elo_x,index_elo_put,3);
        hipDeviceSynchronize();

//        for (int i = 0; i < 15; ++i){
//                printf("%s\n",elo_x[i].ItemId);
//        }
//        printf("SUPER IMPORTANTE %d",index_elo_put);
//        if()

//        pfp_growth(elo_k1,nn,arrayMap, eloMap, ArrayMapSize, eloMapSize);
        index_elo_put=0;
//         run();
       }
}

__global__ void run(EloVector *elo_vetor, ArrayMap *arrayMap,size_t ArrayMapSize) {

    if(threadIdx.x==0){
        printf("%s",elo_vetor[0].eloArray[0].ItemId);
    }
//    if (threadIdx.x < eloMapSize) {
//        pfp_growth(elo_k1,nn,arrayMap, eloMap, ArrayMapSize, eloMapSize);
//    }

}

