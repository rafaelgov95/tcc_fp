#include "hip/hip_runtime.h"
//
// Created by rafael on 20/08/18.
//

#include <cudaHeaders.h>
#include "Kernel.h"
#include "PFPTree.h"
#include "PFPArray.h"
#include <hip/hip_runtime_api.h>
#include <cstdio>
#include "hip/hip_runtime.h"
#include "../include/PFPArray.h"
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
__device__ char * my_strcpy(char *dest, const char *src){
    int i = 0;
    do {
        dest[i] = src[i];}
    while (src[i++] != 0);
    return dest;
}

__device__ char * my_strcat(char *dest, const char *src){
    int i = 0;
    while (dest[i] != 0) i++;
    my_strcpy(dest+i, src);
    return dest;
}
__device__ char * my_cpcat(const char *array1, const char *array2, char *src){
    my_strcat(src,array1);
    my_strcat(src,array2);
    return src;
}
__device__ void reducex_suporte(gpuArrayMap *arrayMap, gpuEloMap *eloMap){
    printf("THREAD FINALIZANDO TRABALHO %d\n",threadIdx.x);
}
__device__ void geracao_candidato(gpuArrayMap *arrayMap, gpuEloMap *eloMap,gpuEloMap **elosMap,int round) {
    auto indexAtual = threadIdx.x;
//    thrust::device_vector<elosMap> B();
    int xxx = 0;
    auto indexParentArrayMap = arrayMap[eloMap[threadIdx.x].indexArrayMap].indexP;
    bool flag = true;
    gpuEloMap *b;
    gpuErrchk(hipMalloc((void **) &b, sizeof(gpuEloMap)*12));
    while (flag) {
        if (arrayMap[indexParentArrayMap].indexP != -1 && arrayMap[indexAtual].indexP != -1) {
            char a[12]="";
            my_cpcat(arrayMap[indexAtual].ItemId,arrayMap[indexParentArrayMap].ItemId, a);

//            my_cpcat(arrayMap[indexAtual].ItemId,arrayMap[indexParentArrayMap].ItemId, b[xxx].ItemId);
            printf("AQUI  %s\n",a);
//            elosMap[threadIdx.x][xxx].indexP=arrayMap[indexParentArrayMap].indexP;
//            elosMap[threadIdx.x][xxx].suporte=arrayMap[indexAtual].suporte;
//            printf("AQUI %s ",b.ItemId);
//            =(gpuArrayMap *)malloc(sizeof(gpuArrayMap));
//            my_cpcat(arrayMap[indexAtual].ItemId,arrayMap[indexParentArrayMap].ItemId,b->ItemId);
//            printf("AQUI  %s\n", my_strcat(arrayMap[indexAtual].ItemId,arrayMap[indexParentArrayMap].ItemId));
//            printf("THEREAD %d ID %s INDEX %d  SUPORTE %d \n", threadIdx.x, elosMap[threadIdx.x][xxx].ItemId,elosMap[threadIdx.x][xxx].indexP,elosMap[threadIdx.x][xxx].suporte);
//                   arrayMap[indexParentArrayMap].ItemId);
        } else {
            flag = false;
        }
        xxx++;
        indexParentArrayMap = arrayMap[eloMap[threadIdx.x + xxx].indexArrayMap].indexP;
    }
    reducex_suporte(arrayMap,eloMap);

}

__global__ void AlgoritmoI(gpuEloMap **elosMaps, gpuArrayMap *arrayMap, gpuEloMap *eloMap, size_t sizeArrayMap,int round) {

    if (threadIdx.x < sizeArrayMap - 1) {
//        printf("ELO ITEM: %s |  INDEX ARRAY %d | SUPORTE %d\n", eloMap[threadIdx.x].ItemId,
//               eloMap[threadIdx.x].indexArrayMap, eloMap[threadIdx.x].suporte);
        geracao_candidato(arrayMap, eloMap,elosMaps,round);

    } else {
        printf("ELO VACOU");
    }


}

__global__ void AlgoritmoI() {
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}
