#include "hip/hip_runtime.h"
//
// Created by rafael on 20/08/18.
//

#include <cudaHeaders.h>
#include "Kernel.h"
#include "PFPTree.h"
#include "PFPArray.h"
#include <hip/hip_runtime_api.h>
#include <cstdio>
#include "hip/hip_runtime.h"
#include "../include/PFPArray.h"
#include "../../../../../../usr/include/form.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

typedef struct {
    Elo elo;
    int size;
}SetMap;

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
__device__ int index_elo_put;


__device__ int compare(char* String_1, char* String_2)
{
    char TempChar_1,
            TempChar_2;

    do
    {
        TempChar_1 = *String_1++;
        TempChar_2 = *String_2++;
    } while(TempChar_1 && TempChar_1 == TempChar_2);

    return TempChar_1 - TempChar_2;
}

__device__ bool my_strcmp( char *array1, char *array2) {
    int i = 0;
    while (array1[i] != '\0') {
        if (array1[i] != array2[i]) {
            return false;
        }
        i++;
    }
    return true;
}
__device__ char *my_strcpy(char *dest, const char *src) {
    int i = 0;
    do {
        dest[i] = src[i];
    } while (src[i++] != 0);
    return dest;
}

__device__ char *my_strcat(char *dest, const char *src) {
    int i = 0;
    while (dest[i] != 0) i++;
    my_strcpy(dest + i, src);
    return dest;
}

__device__ char *my_cpcat(const char *array1, const char *array2, char *src) {
    my_strcat(src, array1);
    my_strcat(src, array2);
    return src;
}

__device__ int index_elo_setmap;
__device__ int index_new_elo;




__global__ void frequencia_x(EloVector *elo_k1,int elo_k1_current,Elo *elo_x,int eloMapSize, int minimo) {
    extern __shared__ SetMap setMap[];
    __shared__ Elo elo_new_put[199];
    auto indexAtual = blockIdx.x * blockDim.x + threadIdx.x; //PC
    bool newFlag = true;
    int indexSetMap = 0;
    int eloSize = 0;
    memset(elo_new_put, 0, sizeof(Elo) * eloMapSize);
    memset(setMap, 0, sizeof(SetMap) * eloMapSize);
    index_elo_setmap =0;
    index_new_elo=0;

    __syncthreads();
        if (threadIdx.x == 0) {
            for (int k = 0; k < eloMapSize; ++k) {
                my_strcpy(setMap[k].elo.ItemId, " ");
            }

        }
    __syncthreads();

        if (indexAtual == 0) {

            for (int k = 0; k < eloMapSize; ++k) {
                int i = 0;
                bool flag = true;
                while (i < eloMapSize && flag) {
                    if (0 == compare(setMap[i].elo.ItemId, " ")) {
                        setMap[i].elo = elo_x[k];
//                        printf("%s %d %d\n", setMap[i].elo.ItemId , setMap[i].elo.indexArrayMap,setMap[i].elo.suporte );
                        elo_k1[elo_k1_current].eloArray[i] = setMap[i].elo;
                        eloSize++;
                        flag = false;
                    } else {
                        if (0 == compare(elo_x[k].ItemId, setMap[i].elo.ItemId)) {
                            flag = false;
                            setMap[i].elo.suporte += elo_x[k].suporte;
                        }
                    }
                    i++;
                }
            }
            atomicAdd(&index_elo_setmap, eloSize);
            elo_k1[elo_k1_current].size = eloSize;
        }
        __syncthreads();
        while (newFlag && indexSetMap < index_elo_setmap) {
            if ((0 == compare(elo_x[indexAtual].ItemId, setMap[indexSetMap].elo.ItemId)) &&
                (setMap[indexSetMap].elo.suporte >= minimo)) {
                elo_new_put[atomicAdd(&index_new_elo, 1)] = elo_x[indexAtual];
                newFlag = false;
            }
            indexSetMap++;
        }
        __syncthreads();

//    if (threadIdx.x == eloMapSize - 1) {
//            for (int i = 0; i < index_new_elo; ++i) {
//                elo_k1[elo_k1_current].eloArray[i] = elo_new_put[i];
//            }
//            elo_k1[elo_k1_current].size = index_new_elo;
//        }

}

__global__ void pfp_growth(EloVector *elo_k1, int *elo_curr ,ArrayMap *arrayMap,size_t arrayMapSize) {
    extern __shared__ Elo elo[];
    auto indexAtual = blockIdx.x * blockDim.x + threadIdx.x;

    int  elo_cur= (*elo_curr)-1;
        int xxx = 0;
        bool flag = true;
        Elo * Elo_k1 = (Elo * )
        malloc(sizeof(Elo) * elo_k1[elo_cur].size);
        while (flag && (indexAtual + xxx) <= elo_k1[elo_cur].size) { //
            auto indexThreadArrayMap = elo_k1[elo_cur].eloArray[indexAtual].indexArrayMap; // indexAtual 13 =  c | parent 4
            auto indexParentArrayMap = arrayMap[elo_k1[elo_cur].eloArray[indexAtual + xxx].indexArrayMap].indexP;
            if (arrayMap[indexThreadArrayMap].indexP != -1 &&
                arrayMap[indexParentArrayMap].indexP != -1) {
                my_cpcat(elo_k1[elo_cur].eloArray[indexAtual].ItemId,
                         arrayMap[indexParentArrayMap].ItemId, Elo_k1[xxx].ItemId);
                Elo_k1[xxx].indexArrayMap = indexParentArrayMap;
                Elo_k1[xxx].suporte = elo_k1[elo_cur].eloArray[indexAtual].suporte;
            } else {
                flag = false;
            }
            xxx++;
        }

// Algoritmo 1 End;
// Algoritmo 2 Begin;

        for (int i = 0; i < (xxx - 1); ++i)
            elo[atomicAdd(&index_elo_put, 1)] = Elo_k1[i];

        if (threadIdx.x == elo_k1[elo_cur].size - 1) {
            Elo * elo_x = (Elo * )
            malloc(sizeof(Elo) * index_elo_put);
            for (int i = 0; i < index_elo_put; ++i) {
                elo_x[i] = elo[i];
//                    printf("INDO PRA MORTE Round :%d  | ELO :%s | IndexArray :%d | Suporte :%d\n",elo_cur,elo_x[i].ItemId,elo_x[i].indexArrayMap,elo_x[i].suporte);
            }


            frequencia_x << < 1, index_elo_put, sizeof(SetMap) * index_elo_put >> >
                                                (elo_k1, elo_cur + 1, elo_x, index_elo_put, 3);
            hipDeviceSynchronize();
            for (int i = 0; i < elo_k1[elo_cur + 1].size; ++i) {
//                printf("VOLTA DA MORTE  Round :%d  | ELO :%s | IndexArray :%d | Suporte :%d\n",elo_cur,elo_k1[elo_cur+1].eloArray[i].ItemId,elo_k1[elo_cur+1].eloArray[i].indexArrayMap,elo_k1[elo_cur+1].eloArray[i].suporte);
            }
            index_elo_put = 0;
            if (elo_k1[elo_cur + 1].size > 0) {
                int x_threads = (elo_k1[elo_cur + 1].size);
                *(elo_curr) = *(elo_curr) + 1;

//                printf("Chamando denovo com %d threads \n", x_threads);
                pfp_growth << < 1, x_threads, elo_k1[elo_cur + 1].size * 4 * sizeof(Elo) >> >
                                              (elo_k1, elo_curr, arrayMap, arrayMapSize);
                hipDeviceSynchronize();

            }
        }

}


