#include "hip/hip_runtime.h"
//
// Created by rafael on 20/08/18.
//

#include <cudaHeaders.h>
#include "Kernel.h"
#include "PFPTree.h"
#include "PFPArray.h"
#include <hip/hip_runtime_api.h>
#include <cstdio>
#include "hip/hip_runtime.h"
#include "../include/PFPArray.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }


inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__device__ char *my_strcpy(char *dest, const char *src) {
    int i = 0;
    do {
        dest[i] = src[i];
    } while (src[i++] != 0);
    return dest;
}

__device__ char *my_strcat(char *dest, const char *src) {
    int i = 0;
    while (dest[i] != 0) i++;
    my_strcpy(dest + i, src);
    return dest;
}

__device__ char *my_cpcat(const char *array1, const char *array2, char *src) {
    my_strcat(src, array1);
    my_strcat(src, array2);
    return src;
}

__device__ void reducex_suporte(gpuEloMap *Elo_k1,gpuArrayMap *arrayMap, gpuEloMap *eloMap, size_t arrayMapSize, size_t eloMapSize,size_t elo_k1_map_size) {

    printf("THREAD %d Elo Size %d\n",  threadIdx.x,elo_k1_map_size);

    for(int i =0;i<elo_k1_map_size;i++)
    printf("THREAD %d CHAR %s\n",  threadIdx.x,Elo_k1[i].ItemId);




//    printf("THREAD FINALIZANDO TRABALHO %d\n", threadIdx.x);
}


__device__ void geracao_candidato( gpuEloMap *Elo_k1,gpuArrayMap *arrayMap, gpuEloMap *eloMap, size_t arrayMapSize, size_t eloMapSize) {
    auto indexAtual = threadIdx.x;
    int xxx = 0;
    auto indexParentArrayMap = arrayMap[eloMap[threadIdx.x].indexArrayMap].indexP;
    bool flag = true;
    while (flag) {
        if (arrayMap[indexParentArrayMap].indexP != -1 && arrayMap[indexAtual].indexP != -1) {
            char a[32] = "";
            my_cpcat(arrayMap[indexAtual].ItemId, arrayMap[indexParentArrayMap].ItemId, a);
            my_strcpy(Elo_k1[xxx].ItemId, a);
            Elo_k1[xxx].indexArrayMap = arrayMap[indexParentArrayMap].indexP;
            Elo_k1[xxx].suporte = arrayMap[indexAtual].suporte;
            printf("THEREAD %d | %s INDEX %d  SUPORTE %d \n", threadIdx.x, Elo_k1[xxx].ItemId,Elo_k1[xxx].indexArrayMap,Elo_k1[xxx].suporte);
        } else {
            flag = false;
        }
        xxx++;
        indexParentArrayMap = arrayMap[eloMap[threadIdx.x + xxx].indexArrayMap].indexP;
    }
    reducex_suporte(Elo_k1,arrayMap,eloMap,arrayMapSize,eloMapSize,(size_t)xxx-1);

}

__global__ void AlgoritmoI(gpuEloMap **Elo_k1, gpuArrayMap *arrayMap, gpuEloMap *eloMap, size_t sizeArrayMap,size_t eloMapSize) {

if (threadIdx.x< sizeArrayMap - 1) {
Elo_k1[threadIdx.x] = (gpuEloMap *)malloc(sizeof(gpuEloMap)*sizeArrayMap);
geracao_candidato(Elo_k1[threadIdx.x],arrayMap,eloMap,sizeArrayMap,eloMapSize);
} else {
printf("ELO VACOU");
}
free(Elo_k1[threadIdx.x]);

}

