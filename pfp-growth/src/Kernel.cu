#include "hip/hip_runtime.h"
//
// Created by rafael on 20/08/18.
//

#include <cudaHeaders.h>
#include "Kernel.h"
#include "PFPTree.h"
#include "PFPArray.h"
#include <hip/hip_runtime_api.h>
#include <cstdio>
#include "hip/hip_runtime.h"
#include "../include/PFPArray.h"

__global__ void AlgoritmoI(gpuArrayMap *v ){
    if(threadIdx.x<12) {
        printf("%d\n", v[threadIdx.x].suporte);
    }
}


__global__ void AlgoritmoI(){
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}
