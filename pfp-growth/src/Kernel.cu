#include "hip/hip_runtime.h"
//
// Created by rafael on 20/08/18.
//

#include <cudaHeaders.h>
#include "Kernel.h"
#include "PFPTree.h"
#include "PFPArray.h"
#include <hip/hip_runtime_api.h>
#include <cstdio>
#include "hip/hip_runtime.h"
#include "../include/PFPArray.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }


inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__device__ char *my_strcpy(char *dest, const char *src) {
    int i = 0;
    do {
        dest[i] = src[i];
    } while (src[i++] != 0);
    return dest;
}

__device__ char *my_strcat(char *dest, const char *src) {
    int i = 0;
    while (dest[i] != 0) i++;
    my_strcpy(dest + i, src);
    return dest;
}

__device__ char *my_cpcat(const char *array1, const char *array2, char *src) {
    my_strcat(src, array1);
    my_strcat(src, array2);
    return src;
}

__device__ void reducex_suporte(gpuEloMap *Elo_k1,gpuArrayMap *arrayMap, gpuEloMap *eloMap, size_t arrayMapSize, size_t eloMapSize,size_t elo_k1_map_size) {



    for(int i =0;i<elo_k1_map_size;i++)
        printf("THREAD %d CHAR %s\n",  threadIdx.x,Elo_k1[i].ItemId);






//    printf("THREAD FINALIZANDO TRABALHO %d\n", threadIdx.x);
}


__device__ void geracao_candidato( gpuEloMap *Elo_k1,gpuArrayMap *arrayMap, gpuEloMap *eloMap, size_t arrayMapSize, size_t eloMapSize) {
    auto indexAtual = threadIdx.x;
    int xxx = 0;
    bool flag = true;
    while (flag) {
        char a[32] = "";
        if(indexAtual+xxx<eloMapSize) {
            auto indexParentArrayMap = arrayMap[eloMap[indexAtual + xxx].indexArrayMap].indexP;
//        printf("THEREAD %d | index %d\n",indexAtual,indexParentArrayMap );
            if (arrayMap[indexParentArrayMap].indexP != -1 &&
                arrayMap[eloMap[indexAtual].indexArrayMap].indexP != -1) {
                my_cpcat(arrayMap[eloMap[indexAtual].indexArrayMap].ItemId,
                         arrayMap[indexParentArrayMap].ItemId, a);
                my_strcpy(Elo_k1[xxx].ItemId, a);
                Elo_k1[xxx].indexArrayMap = arrayMap[indexParentArrayMap].indexP;
                Elo_k1[xxx].suporte = arrayMap[indexAtual].suporte;
                printf("THEREAD %d | xxx %d | %s INDEX %d  SUPORTE %d \n", indexAtual, xxx, Elo_k1[xxx].ItemId,
                       Elo_k1[xxx].indexArrayMap, Elo_k1[xxx].suporte);
            } else {
                my_cpcat(arrayMap[eloMap[indexAtual].indexArrayMap].ItemId,
                         arrayMap[indexParentArrayMap].ItemId, a);
                my_strcpy(Elo_k1[xxx].ItemId, a);
                Elo_k1[xxx].indexArrayMap = arrayMap[indexParentArrayMap].indexP;
                Elo_k1[xxx].suporte = arrayMap[indexAtual].suporte;
//                printf("ERRO NAO ENTRO THEREAD %d | xxx %d | %s INDEX %d  SUPORTE %d \n", indexAtual, xxx,
//                       Elo_k1[xxx].ItemId, Elo_k1[xxx].indexArrayMap, Elo_k1[xxx].suporte);
                flag = false;
            }
            xxx++;
        }else{
            flag = false;
        }
    }
//    reducex_suporte(Elo_k1,arrayMap,eloMap,arrayMapSize,eloMapSize,xxx-1);

}

__global__ void AlgoritmoI(gpuEloMap **Elo_k1, gpuArrayMap *arrayMap, gpuEloMap *eloMap, size_t sizeArrayMap,size_t eloMapSize) {

    if(threadIdx.x< eloMapSize) {

Elo_k1[threadIdx.x] = (gpuEloMap *)malloc(sizeof(gpuEloMap)*eloMapSize);
geracao_candidato(Elo_k1[threadIdx.x],arrayMap,eloMap,sizeArrayMap,eloMapSize);
    }

free(Elo_k1[threadIdx.x]);

}

