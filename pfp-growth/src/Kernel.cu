#include "hip/hip_runtime.h"
//
// Created by rafael on 20/08/18.
//

#include <cudaHeaders.h>
#include "Kernel.h"
#include "PFPTree.h"
#include "PFPArray.h"
#include <hip/hip_runtime_api.h>
#include <cstdio>
#include "hip/hip_runtime.h"
#include "../include/PFPArray.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

typedef struct {
    Elo *elo;
    int *array;
    int size;
}SetMap;

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
__device__ int counter;
__shared__ Elo elo[256];

__device__ int roundd;

__device__ char *my_strcpy(char *dest, const char *src) {
    int i = 0;
    do {
        dest[i] = src[i];
    } while (src[i++] != 0);
    return dest;
}

__device__ char *my_strcat(char *dest, const char *src) {
    int i = 0;
    while (dest[i] != 0) i++;
    my_strcpy(dest + i, src);
    return dest;
}

__device__ char *my_cpcat(const char *array1, const char *array2, char *src) {
    my_strcat(src, array1);
    my_strcat(src, array2);
    return src;
}

//extern __shared__ int s[];

//__device__ void Add_Elo_Frequencia_Elo_k1(Elo *vecor){
//    extern __shared__ int s[];
//}



//    SetMap *setMap =(SetMap*)malloc(sizeof(SetMap));
//    setMap->array=(int*)malloc(sizeof(int)*eloMapSize);
//    setMap->elo=(Elo *)malloc(sizeof(Elo)*eloMapSize);
//    setMap->size=0;
//
//
//    int indexEloMap = eloGrid[threadIdx.x].size;
//    eloGrid[threadIdx.x].eloMap[indexEloMap].elo = Elo_k1;
//    eloGrid[threadIdx.x].eloMap[indexEloMap].size=elo_k1_map_size;
////
//    for (int i = 0; i < eloGrid[threadIdx.x].eloMap[indexEloMap].size; i++) {
//        for (int j = 0;j<setMap->size; ++j) {
//            if(setMap->elo[j].ItemId==eloGrid[threadIdx.x].eloMap[indexEloMap].elo[i].ItemId){
//                setMap->array[j]=setMap->array[j]+1;
//            }
//        }
//        setMap->elo=&eloGrid[threadIdx.x].eloMap[indexEloMap].elo[i];
//        setMap->array[i]=setMap->array[i]+1;
//        setMap->size=setMap->size+1;

//Final_Elo[0] =*
//Elo_k1;
//
__device__ void put_k1_elo(Elo **Elo_k1,Elo *elo_k1, int sizeEloLocal) {
for (int i = 0; i < sizeEloLocal; ++i)
elo[atomicAdd(&roundd,1)]=elo_k1[i];
}

__device__ void reducex_suporte(Elo **Elo_k1,int *sizeArray,Elo *elo_k1, int sizeEloLocal) {


if(threadIdx.x==10){
for (int i = 0; i < roundd; ++i)
printf("%d %s\n",threadIdx.x, elo[i].ItemId);
}



//    int k=0;
//    int kk=(int)nn;
//

//    for (int i=kk; i < (n+kk); ++i) {
//
//        Elo_k1[i] = elo_k1[k];
//        printf("THREAD %d ITEMID %s | IndexArray %d| Suporte %d |\n", threadIdx.x,Elo_k1[i].ItemId,Elo_k1[i].indexArrayMap,Elo_k1[i].suporte);
//
//        k++;
//    }
//    if(threadIdx.x==9){
//        for (int i=0; i <15; ++i) {
//            printf("THREAD %d ITEMID %s | IndexArray %d| Suporte %d |\n", threadIdx.x,Elo_k1[i].ItemId,Elo_k1[i].indexArrayMap,Elo_k1[i].suporte);
//        }
//    }

//    nn=(int*)kk+n;
//    printf("%d %d\n",threadIdx.x ,nn);
//    __syncthreads();
//    if(threadIdx.x==10) {
//        printf("FIM %d %d\n",threadIdx.x ,nn);
//        for (int i = 0; i < nn ; ++i) {
//            printf("THREAD %d ITEMID %s | IndexArray %d| Suporte %d |\n", threadIdx.x,elo_k1[i].ItemId,elo_k1[i].indexArrayMap,elo_k1[i].suporte);
//        }
//    }
//    __syncthreads();

//        printf("THREAD %d ITEMID %s | IndexArray %d| Suporte %d |\n", threadIdx.x,
//               eloGrid[threadIdx.x].eloMap[indexEloMap].elo[i].ItemId,
//               eloGrid[threadIdx.x].eloMap[indexEloMap].elo[i].indexArrayMap,
//               eloGrid[threadIdx.x].eloMap[indexEloMap].elo[i].suporte);
//    }
//    eloGrid[threadIdx.x].size=eloGrid[threadIdx.x].size+1;
////    eloGrid[threadIdx.x].size=eloGrid[threadIdx.x].size+1;
//    for (int k = 0; k < setMap->size ; ++k) {
//        printf("%s | %d\n",setMap->elo[k].ItemId,setMap->array[k]);
//
//    }
}
//for (int i = 0; i < n; ++i){
//printf("THREAD %d ITEMID %s | IndexArray %d| Suporte %d |\n", threadIdx.x, Elo_k1[i].ItemId,
//Elo_k1[i].indexArrayMap,Elo_k1[i].suporte);
//}


__device__ void
geracao_candidato(Elo **elo_k1,int *nn,ArrayMap *arrayMap, Elo *eloMap, size_t arrayMapSize, size_t eloMapSize) {
    auto indexAtual = threadIdx.x;
    int xxx = 0;
    bool flag = true;
//    Elo *elo_k1=(Elo *) malloc(sizeof(Elo*) * eloMapSize);
//    Elo_Grid[threadIdx.x].eloMap[Elo_Grid[threadIdx.x].size].elo = (Elo *) malloc(sizeof(Elo*) * eloMapSize);
//    Elo_Grid[threadIdx.x].eloMap[Elo_Grid[threadIdx.x].size].size =0;

    Elo *Elo_k1 = (Elo *) malloc(sizeof(Elo) * eloMapSize);
    while (flag && (indexAtual + xxx) < eloMapSize) {
        char a[32] = "";
            auto indexParentArrayMap = arrayMap[eloMap[indexAtual + xxx].indexArrayMap].indexP;
            if (arrayMap[indexParentArrayMap].indexP != -1 &&
                arrayMap[eloMap[indexAtual].indexArrayMap].indexP != -1) {
                my_cpcat(arrayMap[eloMap[indexAtual].indexArrayMap].ItemId,
                         arrayMap[indexParentArrayMap].ItemId, a);
                my_strcpy(Elo_k1[xxx].ItemId, a);
                Elo_k1[xxx].indexArrayMap = arrayMap[eloMap[indexAtual+xxx].indexArrayMap].indexP;
                Elo_k1[xxx].suporte = arrayMap[eloMap[indexAtual+xxx].indexArrayMap].suporte;
            } else {
                my_cpcat(arrayMap[eloMap[indexAtual].indexArrayMap].ItemId,
                         arrayMap[indexParentArrayMap].ItemId, a);
                my_strcpy(Elo_k1[xxx].ItemId, a);
                Elo_k1[xxx].indexArrayMap = arrayMap[indexParentArrayMap].indexP;
                Elo_k1[xxx].suporte = arrayMap[indexAtual].suporte;
               flag = false;
            }
            xxx++;

    }
    put_k1_elo(elo_k1,Elo_k1,(xxx-1));
    reducex_suporte(elo_k1,nn,Elo_k1,(xxx - 1));
}

__global__ void run(Elo **Elo_k1,int *nn, ArrayMap *arrayMap, Elo *eloMap, size_t ArrayMapSize, size_t eloMapSize) {

    if (threadIdx.x < eloMapSize) {
    geracao_candidato(Elo_k1,nn,arrayMap, eloMap, ArrayMapSize, eloMapSize);
    }

}

