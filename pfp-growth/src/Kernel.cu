#include "hip/hip_runtime.h"
//
// Created by rafael on 20/08/18.
//

#include <cudaHeaders.h>
#include "Kernel.h"
#include "PFPTree.h"
#include "PFPArray.h"
#include <hip/hip_runtime_api.h>
#include <cstdio>
#include "hip/hip_runtime.h"
#include "../include/PFPArray.h"
#include "../../../../../../usr/include/form.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

typedef struct {
    Elo elo;
    int size;
}SetMap;

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
__device__ char *counter1;
__device__ int inde_new_elo;
__device__ int index_elo_setmap;
__shared__ Elo elo[256];
__device__ int index_elo_put;


__device__ int compare(char* String_1, char* String_2)
{
    char TempChar_1,
            TempChar_2;

    do
    {
        TempChar_1 = *String_1++;
        TempChar_2 = *String_2++;
    } while(TempChar_1 && TempChar_1 == TempChar_2);

    return TempChar_1 - TempChar_2;
}

__device__ bool my_strcmp( char *array1, char *array2) {
    int i = 0;
    while (array1[i] != '\0') {
        if (array1[i] != array2[i]) {
            return false;
        }
        i++;
    }
    return true;
}
__device__ char *my_strcpy(char *dest, const char *src) {
    int i = 0;
    do {
        dest[i] = src[i];
    } while (src[i++] != 0);
    return dest;
}

__device__ char *my_strcat(char *dest, const char *src) {
    int i = 0;
    while (dest[i] != 0) i++;
    my_strcpy(dest + i, src);
    return dest;
}

__device__ char *my_cpcat(const char *array1, const char *array2, char *src) {
    my_strcat(src, array1);
    my_strcat(src, array2);
    return src;
}

__global__ void frequencia_x(EloVector *elo_k1,int elo_k1_current,Elo *elo_x,int eloMapSize, int minimo){
extern __shared__ SetMap setMap[];
 __shared__ Elo elo_new_put[70];
int eloSize=0;


memset(elo_new_put,0,sizeof(Elo)*eloMapSize);

memset(setMap,0,sizeof(SetMap)*eloMapSize);
if(threadIdx.x==0){
   for(int k=0;k<eloMapSize;++k){
          my_strcpy(setMap[k].elo.ItemId," ");
   }

}

__syncthreads();
if(threadIdx.x==0){

for(int k=0;k<eloMapSize;++k){
        int i=0;
        bool flag= true;
            while(i<eloMapSize && flag){
            if(0==compare(setMap[i].elo.ItemId," ")){
                setMap[i].elo=elo_x[k];
                eloSize++;
                flag =false;
            }else{
                if(0==compare(elo_x[k].ItemId,setMap[i].elo.ItemId)){
                    flag =false;
                    setMap[i].elo.suporte+=elo_x[k].suporte;
                    }
            }
            i++;
        }
    }
        atomicAdd(&index_elo_setmap,eloSize);
}
__syncthreads();

bool newFlag=true;
int indexSetMap=0;
while(newFlag && indexSetMap < index_elo_setmap){
    if((0==compare(elo_x[threadIdx.x].ItemId,setMap[indexSetMap].elo.ItemId )) && (setMap[indexSetMap].elo.suporte >= minimo)){
        elo_new_put[atomicAdd(&inde_new_elo,1)]=elo_x[threadIdx.x];
        newFlag=false;
    }
    indexSetMap++;
}


    if(threadIdx.x==eloMapSize-1){
//        for(int i =0;i<index_elo_setmap;++i ){
//            printf("SetMAP Thread %d valor MAP %s Suporte %d \n",threadIdx.x,setMap[i].elo.ItemId,setMap[i].elo.suporte);
//        }
//        for(int i =0;i<inde_new_elo;++i ){
//        printf("Elo_new_PUT Thread %d valor MAP %s Suporte %d \n",threadIdx.x,elo_new_put[i].ItemId,elo_new_put[i].suporte);
//        }
        for (int i = 0; i < index_elo_put; ++i){
                 elo_k1[elo_k1_current].eloArray[i]= elo_new_put[i];
            }
            elo_k1[elo_k1_current].size=index_elo_put;

    }

   }

__device__ void pfp_growth(EloVector *elo_k1, int elo_cur ,ArrayMap *arrayMap,size_t arrayMapSize) {

// Algoritmo 1 Begin;

auto indexAtual = threadIdx.x;
    int xxx = 0;
    bool flag = true;
    Elo *Elo_k1 = (Elo *) malloc(sizeof(Elo) * elo_k1[elo_cur].size);
    while (flag && (indexAtual + xxx) < elo_k1[elo_cur].size) {
            auto indexParentArrayMap = arrayMap[elo_k1[elo_cur].eloArray[indexAtual + xxx].indexArrayMap].indexP;
            auto indexThreadArrayMap = elo_k1[elo_cur].eloArray[indexAtual].indexArrayMap;
            if (arrayMap[indexThreadArrayMap].indexP != -1 &&
                arrayMap[indexParentArrayMap].indexP != -1) {
                my_cpcat(arrayMap[indexThreadArrayMap].ItemId,
                         arrayMap[indexParentArrayMap].ItemId, Elo_k1[xxx].ItemId);
                Elo_k1[xxx].indexArrayMap = arrayMap[indexParentArrayMap].indexP;
                Elo_k1[xxx].suporte = arrayMap[indexThreadArrayMap].suporte;
            } else {
                my_cpcat(arrayMap[indexThreadArrayMap].ItemId,
                         arrayMap[indexParentArrayMap].ItemId, Elo_k1[xxx].ItemId);
                Elo_k1[xxx].indexArrayMap = arrayMap[indexParentArrayMap].indexP;
                Elo_k1[xxx].suporte = arrayMap[indexAtual].suporte;
               flag = false;
            }
            xxx++;

    }
// Algoritmo 1 End;

// Algoritmo 2 Begin;
    for (int i = 0; i < (xxx-1); ++i)
    elo[atomicAdd(&index_elo_put,1)]=Elo_k1[i];

    if (threadIdx.x == elo_k1[elo_cur].size-1 ) {
        Elo *elo_x= (Elo *)malloc(sizeof(Elo)*index_elo_put);
        for (int i = 0; i < index_elo_put; ++i){
                    elo_x[i]= elo[i];
        }
        frequencia_x<<<1,index_elo_put,sizeof(SetMap)*index_elo_put>>>(elo_k1,elo_cur+1,elo_x,index_elo_put,3);
        hipDeviceSynchronize();
        for (int i = 0; i < elo_k1[elo_cur+1].size; ++i){
                printf("%s\n",elo_k1[elo_cur+1].eloArray[i].ItemId);
        }
//        printf("SUPER IMPORTANTE %d",index_elo_put);
//        if()

//        pfp_growth(elo_k1,nn,arrayMap, eloMap, ArrayMapSize, eloMapSize);
        index_elo_put=0;
//         run();
       }
}

__global__ void run(EloVector *elo_vetor, int elo_k1_size,ArrayMap *arrayMap,size_t ArrayMapSize) {
    if (threadIdx.x <  elo_vetor[elo_k1_size].size) {
        pfp_growth(elo_vetor,elo_k1_size,arrayMap, ArrayMapSize);
    }

}

