#include "hip/hip_runtime.h"
//
// Created by rafael on 20/08/18.
//

#include <cudaHeaders.h>
#include <hip/hip_runtime_api.h>
#include <cstdio>
#include "kernel_pfp.h"


__device__ void addNew (int *a, int *b, int *c,int size) {
    int tid = blockIdx.x;
    if (tid < size) c[tid] = a[tid] + b[tid];
}
__global__ void add( int *a, int *b, int *c,int size) {
    addNew(a,b,c,size);

}
    void run(int i)
    {
        int N =i;
        int a[N], b[N], c[N];
        int *dev_a, *dev_b, *dev_c ,*size;
        // allocate the memory on the GPU
          hipMalloc( (void**)&dev_a, N * sizeof(int) ) ;
          hipMalloc( (void**)&dev_b, N * sizeof(int) ) ;
          hipMalloc( (void**)&dev_c, N * sizeof(int) ) ;
          hipMalloc( (void**)&size, sizeof(int) ) ;
         // fill the arrays 'a' and 'b' on the CPU
         for (int i=0; i<N; i++) {
             a[i] = -i;
             b[i] = i * i;
         }
        hipMemcpy( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice );
        hipMemcpy( dev_b, b, N * sizeof(int),hipMemcpyHostToDevice );
        add<<<N,1>>>( dev_a, dev_b, dev_c ,1);
        hipMemcpy( c, dev_c, N * sizeof(int),hipMemcpyDeviceToHost );
        for (int i=0; i<N; i++) {
            printf( "%d + %d = %d\n", a[i], b[i], c[i] );
        }
        hipFree( dev_a );    hipFree( dev_b );    hipFree( dev_c );
    }


