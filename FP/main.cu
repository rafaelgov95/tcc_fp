#include "hip/hip_runtime.h"
#include <iostream>
#include "FPGrowth.h"
#include "FPHeaderTable.h"
#include "FPTransMap.h"
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <algorithm>
#include "numeric"

using namespace cuda_fp_growth;

__global__
void new_header_table( const BitBlock* __restrict__ trans_map, size_type blocks_per_trans,
                       const InnerNode* __restrict__ inner_nodes, const LeafNode* __restrict__ leaf_nodes,
                       const cuda_uint* __restrict__ parent_ht, size_type min_support, index_type node_idx, cuda_uint* output )
{
    FPHeaderTable sub_ht( trans_map, blocks_per_trans, inner_nodes, leaf_nodes, parent_ht, min_support, node_idx );
    hipDeviceSynchronize();
    size_type ht_size = sub_ht.size(), ia_size = sub_ht.ia_size();
    memcpy( output, sub_ht.data(), HTBufferSize( ht_size, ia_size ) );
}

void test_sub_header_table( const FPTransMap& trans_map, const FPRadixTree& radix_tree, const FPHeaderTable& header_table, size_type min_support,
                            index_type node_idx, Items& items, Sizes& counts, Sizes& ia_sizes, Indices& ia_arrays, Sizes& node_counts, NodeTypes& node_types )
{
    thrust::device_vector<cuda_uint> output( HTBufferSize( header_table.size(), header_table.ia_size() ) / sizeof( cuda_uint ), 0 );
    cuda_uint* _output = output.data().get();
    new_header_table <<< 1, 1 >>>( trans_map.bitmap().data().get(), trans_map.blocks_per_transaction(),
            radix_tree.inner_nodes().data().get(), radix_tree.leaf_nodes().data().get(),
            header_table.data(), min_support, node_idx,  _output );
    hipDeviceSynchronize();

    std::vector<cuda_uint> data( output.begin(), output.end() );
    size_type ht_size = data[ 0 ], ia_size = data[ 1 ];
    auto current = data.begin() + 2;

    items.clear();
    items.insert( items.end(), current, current + ht_size );
    current += ht_size;

    counts.clear();
    counts.insert( counts.end(), current, current + ht_size );
    current += ht_size;

    ia_sizes.clear();
    ia_sizes.insert( ia_sizes.end(), current, current + ht_size );
    current += ht_size;

    ia_arrays.clear();
    ia_arrays.insert( ia_arrays.end(), current, current + ht_size * ia_size );
    current += ht_size * ia_size;

    node_counts.clear();
    node_counts.insert( node_counts.end(), current, current + ht_size * ia_size );
    current += ht_size * ia_size;

    node_types.resize( ht_size * ia_size );
    std::transform( current, current + ht_size * ia_size, node_types.begin(), []( cuda_uint value ) { return static_cast<NodeType>( value ); } );
}

void sort_results( const Sizes& ia_sizes, Indices& ia_arrays, Sizes& node_counts, NodeTypes& node_types )
{
    for ( index_type i = 0; i < ia_sizes.size(); ++i ) {
        size_type ia_size = ia_sizes[ i ];
        size_type begin_pos = i * ia_size;

        std::vector<index_type> order( ia_size );
        std::iota( order.begin(), order.end(), begin_pos );
        std::sort( order.begin(), order.end(), [&]( index_type idx_a, index_type idx_b ) {
            NodeType type_a = node_types[ idx_a ], type_b = node_types[ idx_b ];
            index_type ia_a = ia_arrays[ idx_a ], ia_b = ia_arrays[ idx_b ];
            return ( type_a < type_b ) || ( type_a == type_b && ia_a < ia_b );
        } );

        Indices ordered_ia_arrays( ia_size );
        Sizes ordered_node_counts( ia_size );
        NodeTypes ordered_node_types( ia_size );
        for ( index_type i = 0; i < order.size(); ++i ) {
            ordered_ia_arrays[ i ] = ia_arrays[ order[ i ] ];
            ordered_node_counts[ i ] = node_counts[ order[ i ] ];
            ordered_node_types[ i ] = node_types[ order[ i ] ];
        }
        std::move( ordered_ia_arrays.begin(), ordered_ia_arrays.end(), ia_arrays.begin() + begin_pos );
        std::move( ordered_node_counts.begin(), ordered_node_counts.end(), node_counts.begin() + begin_pos );
        std::move( ordered_node_types.begin(), ordered_node_types.end(), node_types.begin() + begin_pos );
    }
}
bool pattern_exists( const std::vector<cuda_uint>& buffer, const std::vector<Item>& pattern, const size_type support, const cuda_real confidence = 0.0f )
{
    index_type i = 0;
    while ( i < buffer.size() ) {
        size_type length = buffer[ i ] / sizeof( cuda_uint );
        size_type offset = ( confidence > 0.0f ? 3 : 2 );
        bool exists = true;
        exists &= ( pattern.size() == length - offset );
        exists &= ( buffer[ i + 1 ] == support );
        exists &= ( std::equal( pattern.begin(), pattern.end(), buffer.begin() + i + offset ) );
        if ( confidence > 0.0f ) {
            const cuda_uint* ptr = &buffer[ i + 2 ];
            exists &= ( std::abs( *( reinterpret_cast<const cuda_real*>( ptr ) ) - confidence ) < 0.0001 );
        }
        if ( exists ) return true;

        i += ( buffer[i] / sizeof( cuda_uint ) );
    }
    return false;
}
size_type pattern_count( const std::vector<cuda_uint>& buffer )
{
    index_type i = 0;
    size_type pattern_count = 0;
    while ( i < buffer.size() ) {
        ++pattern_count;
        i += ( buffer[i] / sizeof( cuda_uint ) );
        std::cout<< "AQUI: "<<( buffer[i] / sizeof( cuda_uint ) ) <<std::endl;
    }


    return pattern_count;
}

int main() {

    const cuda_fp_growth::Item a = 0, b = 1, c = 2, d = 3, e = 4, f = 5, g = 6, h = 7, i = 8, j = 9, k = 10, l = 11, m = 12, n = 13,
            o = 14, p = 15, q = 16, r = 17, s = 18, t = 19, u = 20, v = 21, w = 22, x = 23, y = 24, z = 25;

    cuda_fp_growth::Items trans {
            f, a, c, d, g, i, m, p,
            a, b, c, f, l, m, o,
            b, f, h, j, o,
            b, c, k, s, p,
            a, f, c, e, l, p, m, n
    };



    // start index of each transaction
    Indices indices { 0, 8, 15, 20, 25 };

    // number of items in each transaction
    Sizes sizes { 8, 7, 5, 5, 8 };

    // construct FPTransactionMap object
    size_type min_support = 3;
    FPTransMap trans_map( trans.cbegin(), indices.cbegin(), sizes.cbegin(), indices.size(), min_support );
    FPRadixTree radix_tree( trans_map );
    FPHeaderTable header_table( trans_map, radix_tree, min_support );
    hipDeviceSynchronize();


    if(header_table.size() == 6){
        std::cout<<"FOI"<<std::endl;
    }

    Items items;
    Sizes counts, ia_sizes, node_counts;
    Indices ia_arrays;
    NodeTypes node_types;

    test_sub_header_table( trans_map, radix_tree, header_table, min_support, 0, items, counts, ia_sizes, ia_arrays, node_counts, node_types );
    sort_results( ia_sizes, ia_arrays, node_counts, node_types );
    if( items.size() == 6 ){
        std::cout<<"FOI"<<std::endl;
    };
//
//    REQUIRE( inner_nodes.size() == 3 );
//    CHECK( inner_nodes[ 0 ].parent_idx == 0 );
//    CHECK( inner_nodes[ 0 ].range_start == 0 );
//    CHECK( inner_nodes[ 0 ].range_end == 3 );
//    CHECK( inner_nodes[ 0 ].left_is_leaf == false );
//    CHECK( inner_nodes[ 0 ].right_is_leaf == false );
//    CHECK( inner_nodes[ 0 ].left_idx == 1 );
//    CHECK( inner_nodes[ 0 ].right_idx == 2 );
//    CHECK( inner_nodes[ 0 ].prefix_length == 0 );
//    CHECK( inner_nodes[ 0 ].trans_count == 5 );
//
//    CHECK( inner_nodes[ 1 ].parent_idx == 0 );
//    CHECK( inner_nodes[ 1 ].range_start == 0 );
//    CHECK( inner_nodes[ 1 ].range_end == 1 );
//    CHECK( inner_nodes[ 1 ].left_is_leaf == true );
//    CHECK( inner_nodes[ 1 ].right_is_leaf == true );
//    CHECK( inner_nodes[ 1 ].left_idx == 0 );
//    CHECK( inner_nodes[ 1 ].right_idx == 1 );
//    CHECK( inner_nodes[ 1 ].prefix_length == 3 );
//    CHECK( inner_nodes[ 1 ].trans_count == 2 );
//
//    CHECK( inner_nodes[ 2 ].parent_idx == 0 );
//    CHECK( inner_nodes[ 2 ].range_start == 2 );
//    CHECK( inner_nodes[ 2 ].range_end == 3 );
//    CHECK( inner_nodes[ 2 ].left_is_leaf == true );
//    CHECK( inner_nodes[ 2 ].right_is_leaf == true );
//    CHECK( inner_nodes[ 2 ].left_idx == 2 );
//    CHECK( inner_nodes[ 2 ].right_idx == 3 );
//    CHECK( inner_nodes[ 2 ].prefix_length == 1 );
//    CHECK( inner_nodes[ 2 ].trans_count == 3 );
//
//    REQUIRE( leaf_nodes.size() == 4 );
//    CHECK( leaf_nodes[ 0 ].parent_idx == 1 );
//    CHECK( leaf_nodes[ 0 ].trans_count == 1 );
//
//    CHECK( leaf_nodes[ 1 ].parent_idx == 1 );
//    CHECK( leaf_nodes[ 1 ].trans_count == 1 );
//
//    CHECK( leaf_nodes[ 2 ].parent_idx == 2 );
//    CHECK( leaf_nodes[ 2 ].trans_count == 2 );
//
//    CHECK( leaf_nodes[ 3 ].parent_idx == 2 );
//    CHECK( leaf_nodes[ 3 ].trans_count == 1 );

//    CHECK( pattern_exists( buffer, { a }, 3, 1.0 ) );
//    CHECK( pattern_exists( buffer, { c, a }, 3, 1.0 ) );
//    CHECK( pattern_exists( buffer, { f, a }, 3, 1.0 ) );
//    CHECK( pattern_exists( buffer, { f, c }, 3, 1.0 ) );
//    CHECK( pattern_exists( buffer, { f, c, a }, 3, 1.0 ) );
    return 0;
}
//
/*
   Copyright 2016 Frank Ye

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
 */

//#ifndef UNIT_TEST
//
//// This tells Catch to provide a main() - only do this in one cpp file
//#define CATCH_CONFIG_MAIN
//
//#include "unit-test/catch.hpp"
//
//#else
//
//extern "C"
//{
//}
//
//#endif  // UNIT_TEST